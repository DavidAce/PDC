

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

#define BLOCK_SIZE 256
#define ARRAY_SIZE 16777216

typedef struct timeval tval;

/**
 * Helper method to generate a very naive "hash".
 */
float generate_hash(int n, float *y)
{
    float hash = 0.0f;
    
    for (int i = 0; i < n; i++)
    {
        hash += y[i];
    }
    
    return hash;
}

/**
 * Helper method that calculates the elapsed time between two time intervals (in milliseconds).
 */
long get_elapsed(tval t0, tval t1)
{
    return (t1.tv_sec - t0.tv_sec) * 1000 + (t1.tv_usec - t0.tv_usec) / 1000;
}

/**
 * SAXPY reference implementation using the CPU.
 */
void cpu_saxpy(int n, float a, float *x, float *y)
{
    for (int i = 0; i < n; i++)
    {
        y[i] = a * x[i] + y[i];
    }
}

////////////////
// TO-DO #2.6 /////////////////////////////////////////////////////////////
// Declare the kernel gpu_saxpy() with the same interface as cpu_saxpy() //
///////////////////////////////////////////////////////////////////////////
__global__ void gpu_saxpy(float a, float *x, float *y)
{
    const int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i > ARRAY_SIZE) y[i] = 0 ;
    else  y[i] = a * x[i] + y[i];
}



int compute_num_blocks(){
    int i = ARRAY_SIZE/BLOCK_SIZE;
    int imax = ARRAY_SIZE/BLOCK_SIZE + BLOCK_SIZE;
    while (ARRAY_SIZE % i != 0 and i <= imax ){
        i++;
    }
    return i;
}

int pad_arraysize(){
    int i = ARRAY_SIZE;
    while(i % 32 != 0){
        i++;
    }
    return i;
}


int main(int argc, char **argv)
{
    float a     = 0.0f;
    float *x    = NULL;
    float *y    = NULL;
    float error = 0.0f;

    ////////////////
    // TO-DO #2.2 ///////////////////////////////
    // Introduce the grid and block definition //
    /////////////////////////////////////////////
    const int NUM_BLOCKS = compute_num_blocks();
    dim3 grid(NUM_BLOCKS  ,1,1); // 1 block in the grid
    dim3 block(BLOCK_SIZE,1,1); // 256 threads per block

    printf("grid: %d,  block: %d\n", NUM_BLOCKS, BLOCK_SIZE);

    //////////////////
    // TO-DO #2.3.1 /////////////////////////////
    // Declare the device pointers d_x and d_y //
    /////////////////////////////////////////////
    float *d_x = NULL;
    float *d_y = NULL;
 

    // Make sure the constant is provided
    if (argc != 2)
    {
        fprintf(stderr, "Error: The constant is missing!\n");
        return -1;
    }
    
    // Retrieve the constant and allocate the arrays on the CPU
    a = atof(argv[1]);
    x = (float *)malloc(sizeof(float) * ARRAY_SIZE);
    y = (float *)malloc(sizeof(float) * ARRAY_SIZE);
    
    // Initialize them with fixed values
    for (int i = 0; i < ARRAY_SIZE; i++)
    {
        x[i] = 0.1f;
        y[i] = 0.2f;
    }
    
    //////////////////
    // TO-DO #2.3.2 ////////////////////////////////////////////////////////
    // Allocate d_x and d_y on the GPU, and copy the content from the CPU //
    ////////////////////////////////////////////////////////////////////////
    int D_ARRAY_SIZE = pad_arraysize();
    printf("Actual array size: %d\n", ARRAY_SIZE);
    printf("Padded array size: %d\n", D_ARRAY_SIZE);
    hipMalloc(&d_x, D_ARRAY_SIZE*sizeof(float));
    hipMalloc(&d_y, D_ARRAY_SIZE*sizeof(float));
    hipMemcpy(d_x, x, D_ARRAY_SIZE,hipMemcpyHostToDevice); 
    hipMemcpy(d_y, y, D_ARRAY_SIZE,hipMemcpyHostToDevice);
    tval starttime;
    tval endtime;
    gettimeofday(&starttime,NULL);
    // Call the CPU code
    cpu_saxpy(ARRAY_SIZE, a, x, y);
    gettimeofday(&endtime,NULL);
    int elapsed = get_elapsed(starttime,endtime);
    printf("Time elapsed %d \n", elapsed);

    // Calculate the "hash" of the result from the CPU
    error = generate_hash(ARRAY_SIZE, y);
    
    ////////////////
    // TO-DO #2.4 ////////////////////////////////////////
    // Call the GPU kernel gpu_saxpy() with d_x and d_y //
    //////////////////////////////////////////////////////
    gettimeofday(&starttime,NULL);
    gpu_saxpy<<<NUM_BLOCKS, BLOCK_SIZE>>>(a, d_x,d_y);

    //////////////////
    // TO-DO #2.5.1 ////////////////////////////////////////////////////
    // Copy the content of d_y from the GPU to the array y on the CPU //
    ////////////////////////////////////////////////////////////////////
    
    hipMemcpy(x, d_x, ARRAY_SIZE,hipMemcpyDeviceToHost); 
    hipMemcpy(y, d_y, ARRAY_SIZE,hipMemcpyDeviceToHost);
    gettimeofday(&endtime,NULL);
    elapsed = get_elapsed(starttime,endtime);
    printf("Time elapsed %d \n", elapsed);

    // Calculate the "hash" of the result from the GPU
    error = fabsf(error - generate_hash(D_ARRAY_SIZE, y));
    
    // Confirm that the execution has finished
    printf("Execution finished (error=%.6f).\n", error);
    
    if (error > 0.0001f)
    {
        fprintf(stderr, "Error: The solution is incorrect!\n");
    }
    
    // Release all the allocations
    free(x);
    free(y);
    
    //////////////////
    // TO-DO #2.5.2 /////////
    // Release d_x and d_y //
    /////////////////////////
    hipFree(d_x); // Free the memory
    hipFree(d_y); // Free the memory
    return 0;
}
